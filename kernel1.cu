#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];

    // TODO, implement this kernel below
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

    i = i + 1;
    
    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    j = j + 1;

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) return;

    s_data[blockDim.y * threadIdx.x + (threadIdx.y + 1)] = g_dataA[i * floatpitch + j]; //middle
    s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)] = g_dataA[(i + 1) * floatpitch + j]; //south
    s_data[blockDim.y * (threadIdx.x - 1) + (threadIdx.y + 1)] = g_dataA[(i - 1) * floatpitch + j]; //north

    if(threadIdx.y == blockDim.y - 1)
    {
        s_data[blockDim.y * threadIdx.x + (threadIdx.y + 2)] = g_dataA[ i * floatpitch + (j + 1)]; //E
        s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 2)] = g_dataA[ (i + 1) * floatpitch + (j + 1)]; //SE
        s_data[blockDim.y * (threadIdx.x - 1) + (threadIdx.y + 2)] = g_dataA[ (i - 1) * floatpitch + (j + 1)]; //NE

    }else if(threadIdx.y == 1)
    {
        s_data[blockDim.y * threadIdx.x + threadIdx.y] = g_dataA[ i * floatpitch + (j - 1)]; //W
        s_data[blockDim.y * (threadIdx.x + 1) + threadIdx.y] = g_dataA[ (i + 1) * floatpitch + (j - 1)]; //SW
        s_data[blockDim.y * (threadIdx.x - 1) + threadIdx.y] = g_dataA[ (i - 1) * floatpitch + (j - 1)]; //NW
    }

    __syncthreads();

    float middle = s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)];
    float south = s_data[blockDim.y * (threadIdx.x + 2) + (threadIdx.y + 1)];
    float north = s_data[blockDim.y * threadIdx.x + (threadIdx.y + 1)];

    int colNum = threadIdx.y % 3; // 3 is the number of columns per jacobi value

    float jacobiValue = 1.0;

    //if we are at the first value then we grab the previous value as well otherwise we add up all the values and multiply by .95 to get the jacobi value
    if(colNum == 0)
    {
        s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)] = runJacobi(0, north, middle, south); 

    }else if (colNum == 1)
    {
        s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)] = runJacobi(1, north, middle, south); 
    }else
    {
        jacobiValue = runJacobi(0, north, middle, south);
        jacobiValue += s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y - 1)];
        jacobiValue += s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y - 2)];
        jacobiValue = jacobiValue * 0.95f;
    }

    __syncthreads();

    //if we have a correct jacobiValue, store it at the middle location
    if(colNum == 2 && (blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)) < blockDim.y)
    {
        s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)] = jacobiValue;
    }

    __syncthreads();

    //move data to output array
    if(blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1) < blockDim.y)
        g_dataB[i * floatpitch + j] = s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)];
}

__device__ float runJacobi(int isMiddle, float northValue, float middleValue, float southValue)
{
    if(!isMiddle)
    {
        return 0.1f * northValue + 0.1f * middleValue + 0.1f * southValue;

    }else
    {
        return 0.1f * northValue + 0.2f * middleValue + 0.1f * southValue;
    }

}

