#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];

    // TODO, implement this kernel below
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

    i = i + 1;
    
    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    j = j + 1;

    if(i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) return;

    printf("3");
    if(threadIdx.y != 0)
    {
        s_data[blockDim.y * threadIdx.x + threadIdx.y] = g_dataA[i * floatpitch + j]; //middle
        s_data[blockDim.y * (threadIdx.x + 1) + threadIdx.y] = g_dataA[(i + 1) * floatpitch + j]; //south
        s_data[blockDim.y * (threadIdx.x - 1) + threadIdx.y] = g_dataA[(i - 1) * floatpitch + j]; //north
        
        if(threadIdx.y == blockDim.y - 1 || i == width - 2)
        {
            s_data[blockDim.y * threadIdx.x + (threadIdx.y + 1)] = g_dataA[ i * floatpitch + (j + 1)]; //E
            s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)] = g_dataA[ (i + 1) * floatpitch + (j + 1)]; //SE
            s_data[blockDim.y * (threadIdx.x - 1) + (threadIdx.y + 1)] = g_dataA[ (i - 1) * floatpitch + (j + 1)]; //NE
            printf("0");

        }else if(threadIdx.y == 1)
        {
            s_data[blockDim.y * threadIdx.x + (threadIdx.y - 1)] = g_dataA[ i * floatpitch + (j - 1)]; //W
            s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y - 1)] = g_dataA[ (i + 1) * floatpitch + (j - 1)]; //SW
            s_data[blockDim.y * (threadIdx.x - 1) + (threadIdx.y - 1)] = g_dataA[ (i - 1) * floatpitch + (j - 1)]; //NW
            printf("1");
        }
    }

    __syncthreads();

    if(threadIdx.y > 0 && threadIdx.y < width - 1 && threadIdx.x > 0 && threadIdx.x < width - 1)
    {
        g_dataB[i * floatpitch + j] = (
                              0.2f * s_data[blockDim.y * threadIdx.x + threadIdx.y]              +       //itself
                              0.1f * s_data[blockDim.y * (threadIdx.x - 1) + threadIdx.y]       +       //N
                              0.1f * s_data[blockDim.y * (threadIdx.x - 1) + (threadIdx.y + 1)] +       //NE
                              0.1f * s_data[blockDim.y * threadIdx.x + (threadIdx.y + 1)]       +       //E
                              0.1f * s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y + 1)] +       //SE
                              0.1f * s_data[blockDim.y * (threadIdx.x + 1) + threadIdx.y]       +       //S
                              0.1f * s_data[blockDim.y * (threadIdx.x + 1) + (threadIdx.y - 1)] +       //SW
                              0.1f * s_data[blockDim.y * threadIdx.x + (threadIdx.y - 1)]       +       //W
                              0.1f * s_data[blockDim.y * (threadIdx.x - 1) + (threadIdx.y - 1)]         //NW
                           ) * 0.95f;
    }
}

